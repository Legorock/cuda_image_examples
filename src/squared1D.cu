#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include "squared1D.hpp"

// Kernel function to add the elements of two arrays
__global__
void squared_1D(float* input_image, float* output_image, int image_size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = idx; i < image_size ; i += gridDim.x) {
        output_image[i] = input_image[i] * input_image[i];
    }
}

GrayScaleImage<float> squared1D(const GrayScaleImage<float>& image)
{
    int image_size = image.height * image.width;
    float *input_image, *output_image;

    std::cout << "squared1D with size:" << image_size << std::endl;

    // Allocate Unified Memory – accessible from CPU or GPU
    auto err = hipMallocManaged(&input_image, image_size*sizeof(float));
    if (err != 0) { std::cerr << "Cuda malloc error: " << err << std::endl; }
    err = hipMallocManaged(&output_image, image_size*sizeof(float));
    if (err != 0) { std::cerr << "Cuda malloc error: " << err << std::endl; }

    // initialize and copy arrays on the host
    for (int i = 0; i < image_size; i++) {
        // input_image[i] = image.pixels[i];
        // output_image[i] = 0;
    }

    std::cout << "squared1D before launch" << std::endl;

    int num_threads = 1024;
    int num_blocks = std::max(image_size / num_threads, 1024);

    // squared_1D<<<num_blocks, num_threads>>>(input_image, output_image, image_size);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // copy back to CPU
    GrayScaleImage<float> output;
    output.height = image.height;
    output.width = image.width;
    output.pixels = std::unique_ptr<float[]>(new float[image_size]);

    for (int i = 0; i < image_size; i++) {
        output.pixels[i] = output_image[i];
    }

    std::cout << "squared1D before free" << std::endl;

    hipFree(input_image);
    hipFree(output_image);

    return output;
}
